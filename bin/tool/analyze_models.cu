#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <map>
#include <filesystem>
#include <thread>
#include <mutex>

#include <BS_thread_pool.hpp>

#include "cuda_util.hpp"

constexpr uint32_t BLOCK_SIZE = 1024;
template <typename T>
__global__ void calculate_square_of_value(T* output, T* input0, T* input1, uint32_t size)
{
    uint32_t block_x = blockIdx.x;
    uint32_t thread_x = threadIdx.x;
    uint32_t index = block_x * BLOCK_SIZE + thread_x;

    if (index >= size) return;
    auto diff = (input0[index] - input1[index]);
    output[index] = diff * diff;
}

std::vector<float> calculate_square_of_value_host(const std::vector<float>& input0, const std::vector<float>& input1)
{
    uint32_t output_size = input0.size();
    if (output_size > input1.size())
    {
        output_size = input1.size();
    }
    uint32_t input0_size = input0.size();
    uint32_t input1_size = input1.size();

    uint32_t block_count = output_size / BLOCK_SIZE;

    float* input0_dev;
    float* input1_dev;
    float* output_dev;
    hipStream_t hip_stream;
    checkCudaErrors(hipStreamCreate(&hip_stream));
    checkCudaErrors(hipMalloc((void **)&input0_dev, input0_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&input1_dev, input1_size * sizeof(float)));
    checkCudaErrors(hipMalloc((void **)&output_dev, output_size * sizeof(float)));
    checkCudaErrors(hipMemcpy(input0_dev, input0.data(), input0_size * sizeof(float), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(input1_dev, input1.data(), input1_size * sizeof(float), hipMemcpyHostToDevice));

    calculate_square_of_value<float><<<block_count, BLOCK_SIZE, 0, hip_stream>>>(output_dev, input0_dev, input1_dev, output_size);
    getLastCudaError("fail to start kernel <<<calculate_square_of_value>>>");
    checkCudaErrors(hipStreamSynchronize(hip_stream));

    std::vector<float> output;
    output.resize(output_size);
    checkCudaErrors(hipMemcpy(output.data(), output_dev, output_size * sizeof(float), hipMemcpyDeviceToHost));
    return output;
}

class cuda_stream_manager
{
public:
    explicit cuda_stream_manager(size_t stream_count)
    {
        _cuda_streams.resize(stream_count);
        for (auto& stream: _cuda_streams)
        {
            hipStreamCreate(&stream);
        }
    }
    
    hipStream_t get_cuda_stream()
    {
        stream_index++;
        if (stream_index >= _cuda_streams.size()) stream_index = 0;
        return _cuda_streams[stream_index];
    }
    
    void all_stream_synchronize()
    {
        for (auto& stream: _cuda_streams)
        {
            checkCudaErrors(hipStreamSynchronize(stream));
        }
    }
    
    size_t get_manager_stream_count()
    {
        return _cuda_streams.size();
    }
    
private:
    int stream_index = 0;
    std::vector<hipStream_t> _cuda_streams;
};

static cuda_stream_manager static_cuda_stream_manager(16);

std::map<std::pair<std::string, std::string>, std::map<std::string, float>> calculate_model_distance_of_each_model_pair_gpu_kernel(const std::map<std::string, std::map<std::string, std::vector<float>>>& node_layer_weight)
{
    std::map<std::pair<std::string, std::string>, std::map<std::string, float>> output;
    std::mutex output_lck;
    
    //allocate output
    for (auto iter_l = node_layer_weight.begin(); iter_l != node_layer_weight.end() ; ++iter_l)
    {
        for (auto iter_r = iter_l; iter_r != node_layer_weight.end(); ++iter_r)
        {
            if (iter_r == iter_l) continue;
            for (const auto &[layer_name, weight_l]: iter_l->second)
            {
                auto node_pair = std::make_pair(iter_l->first, iter_r->first);
                output[node_pair][layer_name] = 0;
            }
        }
    }
    
    std::map<std::string, float*> node_layer_to_device_memory;
    
    //copy layer weight to GPU
    {
        auto cu_stream = static_cuda_stream_manager.get_cuda_stream();
        for (const auto& [node_name, layer_weight] : node_layer_weight)
        {
            for (const auto& [layer, weight] : layer_weight)
            {
                float* temp_device_ptr;
                checkCudaErrors(hipMallocAsync((void **)&temp_device_ptr, weight.size() * sizeof(weight[0]), cu_stream));
                checkCudaErrors(hipMemcpyAsync(temp_device_ptr, weight.data(), weight.size() * sizeof(weight[0]), hipMemcpyHostToDevice, cu_stream));
                node_layer_to_device_memory.emplace(node_name+layer, temp_device_ptr);
            }
        }
        checkCudaErrors(hipStreamSynchronize(cu_stream));
    }

    
    std::vector<std::thread> pools;
    for (auto iter_l = node_layer_weight.begin(); iter_l != node_layer_weight.end() ; ++iter_l)
    {
        for (auto iter_r = iter_l; iter_r != node_layer_weight.end(); ++iter_r)
        {
            if (iter_r == iter_l) continue;
            
            for (const auto& [layer_name, weight_l] : iter_l->second)
            {
                auto hip_stream = static_cuda_stream_manager.get_cuda_stream();

                std::thread temp_thread([iter_l, iter_r, &node_layer_to_device_memory, &output, &hip_stream, &output_lck, layer_name, weight_l](){
                    auto lhs_device_data_iter = node_layer_to_device_memory.find(iter_l->first + layer_name);
                    if (lhs_device_data_iter == node_layer_to_device_memory.end()) throw std::logic_error("logic_error");
                    float* lhs_device_data = lhs_device_data_iter->second;
                    
                    auto rhs_device_data_iter = node_layer_to_device_memory.find(iter_r->first + layer_name);
                    if (rhs_device_data_iter == node_layer_to_device_memory.end()) throw std::logic_error("logic_error");
                    float* rhs_device_data = rhs_device_data_iter->second;
                    
                    auto output_size_bit = weight_l.size() * sizeof(weight_l[0]);
                    
                    //allocate output region on gpu
                    float* output_dev;
                    checkCudaErrors(hipMallocAsync((void **)&output_dev, output_size_bit, hip_stream));
                    
                    size_t data_size = weight_l.size();
                    uint32_t block_count = data_size / BLOCK_SIZE + 1;
                    calculate_square_of_value<float><<<block_count, BLOCK_SIZE, 0, hip_stream>>>(output_dev, lhs_device_data, rhs_device_data, weight_l.size());
                    getLastCudaError("fail to start kernel <<<calculate_square_of_value>>>");
                    
                    //copy back to host
                    std::vector<float> host_buffer;
                    host_buffer.resize(weight_l.size());
                    checkCudaErrors(hipMemcpyAsync(host_buffer.data(), output_dev, output_size_bit, hipMemcpyDeviceToHost, hip_stream));
                    checkCudaErrors(hipFreeAsync(output_dev, hip_stream));
                    checkCudaErrors(hipStreamSynchronize(hip_stream));
                    
                    float v = 0;
                    for (const auto& i: host_buffer)
                    {
                        v += i;
                    }
                    auto node_pair = std::make_pair(iter_l->first, iter_r->first);
                    
                    {
                        std::lock_guard guard(output_lck);
                        output.at(node_pair).at(layer_name) = std::sqrt(v);
                    }
                });
                
                std::thread dummy;
                dummy.swap(temp_thread);
                pools.push_back(std::move(dummy));
            }
        }
    }
    for (auto& thread: pools)
    {
        thread.join();
    }
    
    static_cuda_stream_manager.all_stream_synchronize();
    
    //clear gpu memory
    {
        auto cu_stream = static_cuda_stream_manager.get_cuda_stream();
        for (const auto& [_, device_memory] : node_layer_to_device_memory)
        {
            checkCudaErrors(hipFreeAsync(device_memory, cu_stream));
        }
        checkCudaErrors(hipStreamSynchronize(cu_stream));
    }

    
    return output;
}
