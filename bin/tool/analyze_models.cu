#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <map>
#include <filesystem>
#include <thread>
#include <mutex>

#include <BS_thread_pool.hpp>

#include "cuda_util.hpp"

constexpr uint32_t BLOCK_SIZE = 1024;
template <typename T>
__global__ void calculate_square_of_value(T* output, T* input0, T* input1, uint32_t size)
{
    uint32_t block_x = blockIdx.x;
    uint32_t thread_x = threadIdx.x;
    uint32_t index = block_x * BLOCK_SIZE + thread_x;

    if (index >= size) return;
    auto diff = (input0[index] - input1[index]);
    output[index] = diff * diff;
}

class cuda_stream_manager
{
private:
    bool _device_support_async_mem_management;
    
public:
    bool get_device_support_async_mem_management() const
    {
        return _device_support_async_mem_management;
    }
    
    explicit cuda_stream_manager(size_t stream_count)
    {
        _cuda_streams.resize(stream_count);
        for (auto& stream: _cuda_streams)
        {
            checkCudaErrors(hipStreamCreate(&stream));
        }
        int attr = 0, device = 0;
        hipGetDevice(&device);
        hipDeviceGetAttribute(&attr, hipDeviceAttributeMemoryPoolsSupported, device);
        if (attr == 1) _device_support_async_mem_management = true;
        else _device_support_async_mem_management = false;
    }
    
    hipStream_t get_cuda_stream()
    {
        stream_index++;
        if (stream_index >= _cuda_streams.size()) stream_index = 0;
        return _cuda_streams[stream_index];
    }
    
    void all_stream_synchronize()
    {
        for (auto& stream: _cuda_streams)
        {
            checkCudaErrors(hipStreamSynchronize(stream));
        }
    }
    
    size_t get_manager_stream_count()
    {
        return _cuda_streams.size();
    }
    
private:
    int stream_index = 0;
    std::vector<hipStream_t> _cuda_streams;
};

static cuda_stream_manager static_cuda_stream_manager(4);

void sync_all_cuda_stream()
{
    static_cuda_stream_manager.all_stream_synchronize();
    checkCudaErrors(hipStreamSynchronize(hipStream_t(0)));
}

void allocate_and_copy_device_memory(float** temp_device_ptr, const float* host_data, size_t size)
{
    if (static_cuda_stream_manager.get_device_support_async_mem_management())
    {
        checkCudaErrors(hipMallocAsync((void **)temp_device_ptr, size, hipStream_t(0)));
        checkCudaErrors(hipMemcpyAsync(*temp_device_ptr, host_data, size, hipMemcpyHostToDevice, hipStream_t(0)));
        checkCudaErrors(hipStreamSynchronize(hipStream_t(0)));
    }
    else
    {
        checkCudaErrors(hipMalloc((void **)temp_device_ptr, size));
        checkCudaErrors(hipMemcpy(*temp_device_ptr, host_data, size, hipMemcpyHostToDevice));
        checkCudaErrors(hipStreamSynchronize(hipStream_t(0)));
    }
}

std::vector<float> run_kernel(const std::vector<float>& weight_l, float* lhs_device_data, float* rhs_device_data)
{
    auto hip_stream = static_cuda_stream_manager.get_cuda_stream();
    
    //allocate output region on gpu
    auto output_size_bit = weight_l.size() * sizeof(weight_l[0]);
    float* output_dev;
    
    if (static_cuda_stream_manager.get_device_support_async_mem_management())
    {
        checkCudaErrors(hipMallocAsync((void **)&output_dev, output_size_bit, hip_stream));
    }
    else
    {
        checkCudaErrors(hipMalloc((void **)&output_dev, output_size_bit));
    }
    
    size_t data_size = weight_l.size();
    uint32_t block_count = data_size / BLOCK_SIZE + 1;
    if (static_cuda_stream_manager.get_device_support_async_mem_management())
    {
        calculate_square_of_value<float><<<block_count, BLOCK_SIZE, 0, hip_stream>>>(output_dev, lhs_device_data, rhs_device_data, weight_l.size());
    }
    else
    {
        calculate_square_of_value<float><<<block_count, BLOCK_SIZE>>>(output_dev, lhs_device_data, rhs_device_data, weight_l.size());
    }

    getLastCudaError("fail to start kernel <<<calculate_square_of_value>>>");

    //copy back to host
    std::vector<float> host_buffer;
    host_buffer.resize(weight_l.size());
    
    if (static_cuda_stream_manager.get_device_support_async_mem_management())
    {
        checkCudaErrors(hipMemcpyAsync(host_buffer.data(), output_dev, output_size_bit, hipMemcpyDeviceToHost, hip_stream));
        checkCudaErrors(hipFreeAsync(output_dev, hip_stream));
        checkCudaErrors(hipStreamSynchronize(hip_stream));
    }
    else
    {
        checkCudaErrors(hipMemcpy(host_buffer.data(), output_dev, output_size_bit, hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(output_dev));
        checkCudaErrors(hipStreamSynchronize(hipStream_t(0)));
    }
    
    return host_buffer;
}

void clear_gpu_memory(const std::map<std::string, float*>& node_layer_to_device_memory)
{
    for (const auto& [_, device_memory] : node_layer_to_device_memory)
    {
        if (static_cuda_stream_manager.get_device_support_async_mem_management())
        {
            checkCudaErrors(hipFreeAsync(device_memory, hipStream_t(0)));
        }
        else
        {
            checkCudaErrors(hipFree(device_memory));
        }
    }
}

