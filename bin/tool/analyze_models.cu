#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <vector>
#include <map>
#include <filesystem>
#include <thread>
#include <mutex>

#include <BS_thread_pool.hpp>

#include "cuda_util.hpp"

constexpr uint32_t BLOCK_SIZE = 1024;
template <typename T>
__global__ void calculate_square_of_value(T* output, T* input0, T* input1, uint32_t size)
{
    uint32_t block_x = blockIdx.x;
    uint32_t thread_x = threadIdx.x;
    uint32_t index = block_x * BLOCK_SIZE + thread_x;

    if (index >= size) return;
    auto diff = (input0[index] - input1[index]);
    output[index] = diff * diff;
}

class cuda_stream_manager
{
public:
    explicit cuda_stream_manager(size_t stream_count)
    {
        _cuda_streams.resize(stream_count);
        for (auto& stream: _cuda_streams)
        {
            checkCudaErrors(hipStreamCreate(&stream));
        }
    }
    
    hipStream_t get_cuda_stream()
    {
        stream_index++;
        if (stream_index >= _cuda_streams.size()) stream_index = 0;
        return _cuda_streams[stream_index];
    }
    
    void all_stream_synchronize()
    {
        for (auto& stream: _cuda_streams)
        {
            checkCudaErrors(hipStreamSynchronize(stream));
        }
    }
    
    size_t get_manager_stream_count()
    {
        return _cuda_streams.size();
    }
    
private:
    int stream_index = 0;
    std::vector<hipStream_t> _cuda_streams;
};

static cuda_stream_manager static_cuda_stream_manager(8);

void sync_all_cuda_stream()
{
    static_cuda_stream_manager.all_stream_synchronize();
    checkCudaErrors(hipStreamSynchronize(hipStream_t(0)));
}

void allocate_and_copy_device_memory(float** temp_device_ptr, const float* host_data, size_t size)
{
#if ANALYZE_MODEL_USE_SINGLE_CUDA_STREAM
    checkCudaErrors(hipMalloc((void **)temp_device_ptr, size));
    checkCudaErrors(hipMemcpy(*temp_device_ptr, host_data, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipStreamSynchronize(hipStream_t(0)));
#else
    checkCudaErrors(hipMallocAsync((void **)temp_device_ptr, size, hipStream_t(0)));
    checkCudaErrors(hipMemcpyAsync(*temp_device_ptr, host_data, size, hipMemcpyHostToDevice, hipStream_t(0)));
    checkCudaErrors(hipStreamSynchronize(hipStream_t(0)));
#endif
}

std::vector<float> run_kernel(const std::vector<float>& weight_l, float* lhs_device_data, float* rhs_device_data)
{
#if not ANALYZE_MODEL_USE_SINGLE_CUDA_STREAM
    auto hip_stream = static_cuda_stream_manager.get_cuda_stream();
#endif
    
    //allocate output region on gpu
    auto output_size_bit = weight_l.size() * sizeof(weight_l[0]);
    float* output_dev;

#if ANALYZE_MODEL_USE_SINGLE_CUDA_STREAM
    checkCudaErrors(hipMalloc((void **)&output_dev, output_size_bit));
#else
    checkCudaErrors(hipMallocAsync((void **)&output_dev, output_size_bit, hip_stream));
#endif

    
    size_t data_size = weight_l.size();
    uint32_t block_count = data_size / BLOCK_SIZE + 1;
#if ANALYZE_MODEL_USE_SINGLE_CUDA_STREAM
    calculate_square_of_value<float><<<block_count, BLOCK_SIZE>>>(output_dev, lhs_device_data, rhs_device_data, weight_l.size());
#else
    calculate_square_of_value<float><<<block_count, BLOCK_SIZE, 0, hip_stream>>>(output_dev, lhs_device_data, rhs_device_data, weight_l.size());
#endif

    getLastCudaError("fail to start kernel <<<calculate_square_of_value>>>");

    //copy back to host
    std::vector<float> host_buffer;
    host_buffer.resize(weight_l.size());
#if ANALYZE_MODEL_USE_SINGLE_CUDA_STREAM
    checkCudaErrors(hipMemcpy(host_buffer.data(), output_dev, output_size_bit, hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(output_dev));
    checkCudaErrors(hipStreamSynchronize(hipStream_t(0)));
#else
    checkCudaErrors(hipMemcpyAsync(host_buffer.data(), output_dev, output_size_bit, hipMemcpyDeviceToHost, hip_stream));
    checkCudaErrors(hipFreeAsync(output_dev, hip_stream));
    checkCudaErrors(hipStreamSynchronize(hip_stream));
#endif
    
    return host_buffer;
}

void clear_gpu_memory(const std::map<std::string, float*>& node_layer_to_device_memory)
{
    for (const auto& [_, device_memory] : node_layer_to_device_memory)
    {
#if ANALYZE_MODEL_USE_SINGLE_CUDA_STREAM
        checkCudaErrors(hipFree(device_memory));
#else
        checkCudaErrors(hipFreeAsync(device_memory, hipStream_t(0)));
#endif

    }
}

